#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <string>
#include <chrono>

#include "poly_mul.hpp"

void test_correctness(int block_size = 256) {
    auto generate_polynomial = [](int degree) {
        static auto rd = std::random_device {};
        static auto gen = std::mt19937(rd());
        static auto dis = std::uniform_int_distribution<int>(-1, 1);
        auto poly = std::vector<int>(degree + 1, 1);
        for (int i = 0; i < degree + 1; ++i) {
            poly[i] = dis(gen);
        }
        return poly;
    };

    std::cout << "Running correctness tests...\n";
    auto const test_degrees = std::vector {
        1, 2, 4, 8, 
        16, 32, 64, 128, 
        256, 512, 1024, 2048, 
        (1 << 16)
    };
    for (int const degree : test_degrees) {
        auto poly1 = generate_polynomial(degree);
        auto poly2 = generate_polynomial(degree);
        auto result = std::vector<int>(2 * degree + 1);
        auto expected = std::vector<int>(2 * degree + 1);

        // serial code on CPU for testing
        naive_polynomial_mul_cpu(poly1.data(), poly2.data(), expected.data(), degree);
        // polynomial CUDA kernel
        polynomial_mul(poly1.data(), poly2.data(), result.data(), degree, block_size, PolynomialMulMethod::fast);

        if (result != expected) {
            std::cerr << "[FAIL] Test failed for degree " << degree
                << "\nPolynomial 1: ";
            for (auto const coeff : poly1) std::cerr << coeff << " ";
            std::cerr << "\nPolynomial 2: ";
            for (auto const coeff : poly2) std::cerr << coeff << " ";
            std::cerr << "\nExpected: ";
            for (auto const coeff : expected) std::cerr << coeff << " ";
            std::cerr << "\nGot: ";
            for (auto const coeff : result) std::cerr << coeff << " ";
            std::cerr << '\n';
            return;
        }

        std::cout << "[PASS] Test passed for degree " << degree << "\n";
    }
    std::cout << "\nPASSED ALL TESTS\n";
}

void test_naive_performance(int degree) {
    const int numCoefficients = 2 * degree + 1;

    // Define polynomials
    int poly1[numCoefficients];
    int poly2[numCoefficients];
    std::fill(poly1, poly1 + numCoefficients, 1.0f);
    std::fill(poly2, poly2 + numCoefficients, 1.0f);
    int result[numCoefficients];

    // use std::chrono to measure time it takes to run naive_polynomial_mul
    auto start = std::chrono::high_resolution_clock::now();
    naive_polynomial_mul_cpu(poly1, poly2, result, degree);
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    auto elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed);

    // print degree, block size, and elapsed time
    std::cout << "[cpu] degree: " << degree
        << ", time: " << elapsed_ms.count() << " ms\n";

}

void test_performance(
    int degree, 
    int block_size,
    PolynomialMulMethod method = PolynomialMulMethod::fast)
{
    const int numCoefficients = 2 * degree + 1;

    // Define polynomials
    int poly1[numCoefficients];
    int poly2[numCoefficients];
    std::fill(poly1, poly1 + numCoefficients, 1.0f);
    std::fill(poly2, poly2 + numCoefficients, 1.0f);
    int result[numCoefficients];

    // Create CUDA events for profiling
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Perform polynomial multiplication using the given method
    hipEventRecord(start); // Record start time
    polynomial_mul(poly1, poly2, result, degree, block_size, method);
    hipEventRecord(stop); // Record stop time

    // Synchronize to make sure all CUDA operations are completed
    hipDeviceSynchronize();

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print result
    std::string method_name;
    switch (method) {
        case PolynomialMulMethod::naive:
            method_name = "naive";
            break;
        case PolynomialMulMethod::fast:
            method_name = "fast!";
            break;
    }

    // print method name, degree, block size, and elapsed time
    std::cout << "[cuda (" << method_name << ")] degree: " << degree
        << ", block size: " << block_size
        << ", time: " << milliseconds << " ms\n";

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);

}


int main() {
    test_correctness();

    for (int i = 1; i <= 16; i++) {
        test_naive_performance(1 << i);
    }

    for (int b = 32; b <= 512; b *= 2) {
        for (int i = 1; i <= 16; i++) {
            test_performance(1 << i, b, PolynomialMulMethod::naive);
            test_performance(1 << i, b, PolynomialMulMethod::fast);
        }
    }
    return 0;
}
